#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/extrema.h>

#include "batch_factorize.h"
#include <cstdio>
#include <vector>
#include <iostream>


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Marshalling routines for batched execution 
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<class T>
inline void marshallBatchedLUData(TBatchFactorizeWorkspace<T>* ws, int_t k_st, int_t k_end)
{
    TBatchLUMarshallData<T>& mdata = ws->marshall_data;
    LocalLU_type<T>& d_localLU = ws->d_localLU;

    mdata.batchsize = k_end - k_st;

    TMarshallLUFunc<T> func(
        k_st, mdata.dev_diag_ptrs, mdata.dev_diag_ld_array, mdata.dev_diag_dim_array, 
        d_localLU.Lnzval_bc_ptr, d_localLU.Lrowind_bc_ptr, ws->perm_c_supno, ws->xsup
    );

    thrust::for_each(
        thrust::system::cuda::par, thrust::counting_iterator<int_t>(0),
        thrust::counting_iterator<int_t>(mdata.batchsize), func
    );
}

template<class T>
inline void marshallBatchedTRSMUData(TBatchFactorizeWorkspace<T>* ws, int_t k_st, int_t k_end)
{
    TBatchLUMarshallData<T>& mdata = ws->marshall_data;
    LocalLU_type<T>& d_localLU = ws->d_localLU;

    mdata.batchsize = k_end - k_st;

    TMarshallTRSMUFunc<T> func(
        k_st, mdata.dev_diag_ptrs, mdata.dev_diag_ld_array, mdata.dev_diag_dim_array, 
        mdata.dev_panel_ptrs, mdata.dev_panel_ld_array, mdata.dev_panel_dim_array, 
        d_localLU.Unzval_br_new_ptr, d_localLU.Ucolind_br_ptr, d_localLU.Lnzval_bc_ptr, 
        d_localLU.Lrowind_bc_ptr, ws->perm_c_supno, ws->xsup
    );

    thrust::for_each(
        thrust::system::cuda::par, thrust::counting_iterator<int_t>(0),
        thrust::counting_iterator<int_t>(mdata.batchsize), func
    );
}

template<class T>
inline void marshallBatchedTRSMLData(TBatchFactorizeWorkspace<T>* ws, int_t k_st, int_t k_end)
{
    TBatchLUMarshallData<T>& mdata = ws->marshall_data;
    LocalLU_type<T>& d_localLU = ws->d_localLU;
    
    mdata.batchsize = k_end - k_st;

    TMarshallTRSMLFunc<T> func(
        k_st, mdata.dev_diag_ptrs, mdata.dev_diag_ld_array, mdata.dev_diag_dim_array, 
        mdata.dev_panel_ptrs, mdata.dev_panel_ld_array, mdata.dev_panel_dim_array,
        d_localLU.Lnzval_bc_ptr, d_localLU.Lrowind_bc_ptr, ws->perm_c_supno, ws->xsup
    );

    thrust::for_each(
        thrust::system::cuda::par, thrust::counting_iterator<int_t>(0),
        thrust::counting_iterator<int_t>(mdata.batchsize), func
    );
}

template<class T>
inline void marshallBatchedSCUData(TBatchFactorizeWorkspace<T>* ws, int_t k_st, int_t k_end)
{
    TBatchSCUMarshallData<T>& sc_mdata = ws->sc_marshall_data;
    LocalLU_type<T>& d_localLU = ws->d_localLU;

    sc_mdata.batchsize = k_end - k_st;
    
    thrust::counting_iterator<int_t> start(0), end(sc_mdata.batchsize);
    
    TMarshallSCUFunc<T> func(
        k_st, sc_mdata.dev_A_ptrs, sc_mdata.dev_lda_array, sc_mdata.dev_B_ptrs, sc_mdata.dev_ldb_array, 
        sc_mdata.dev_C_ptrs, sc_mdata.dev_ldc_array, sc_mdata.dev_m_array, sc_mdata.dev_n_array, sc_mdata.dev_k_array,
        sc_mdata.dev_ist, sc_mdata.dev_iend, sc_mdata.dev_jst, sc_mdata.dev_jend, d_localLU.Unzval_br_new_ptr, d_localLU.Ucolind_br_ptr, 
        d_localLU.Lnzval_bc_ptr, d_localLU.Lrowind_bc_ptr, ws->perm_c_supno, ws->xsup, ws->gemm_buff_ptrs
    );

    thrust::for_each(thrust::system::cuda::par, start, end, func);

    // Set the max dims in the marshalled data 
    sc_mdata.max_m = thrust::reduce(thrust::system::cuda::par, sc_mdata.dev_m_array, sc_mdata.dev_m_array + sc_mdata.batchsize, 0, thrust::maximum<BatchDim_t>());
    sc_mdata.max_n = thrust::reduce(thrust::system::cuda::par, sc_mdata.dev_n_array, sc_mdata.dev_n_array + sc_mdata.batchsize, 0, thrust::maximum<BatchDim_t>());
    sc_mdata.max_k = thrust::reduce(thrust::system::cuda::par, sc_mdata.dev_k_array, sc_mdata.dev_k_array + sc_mdata.batchsize, 0, thrust::maximum<BatchDim_t>());
    sc_mdata.max_ilen = thrust::transform_reduce(thrust::system::cuda::par, start, end, element_diff<BatchDim_t>(sc_mdata.dev_ist, sc_mdata.dev_iend), 0, thrust::maximum<BatchDim_t>());
    sc_mdata.max_jlen = thrust::transform_reduce(thrust::system::cuda::par, start, end, element_diff<BatchDim_t>(sc_mdata.dev_jst, sc_mdata.dev_jend), 0, thrust::maximum<BatchDim_t>());
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Utility routines
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
struct BatchLDataSizeAssign_Func {
    int_t** Lrowind_bc_ptr;
    int64_t *d_lblock_gid_offsets, *d_lblock_start_offsets;

    BatchLDataSizeAssign_Func(int_t** Lrowind_bc_ptr, int64_t* d_lblock_gid_offsets, int64_t* d_lblock_start_offsets)
    {
        this->Lrowind_bc_ptr = Lrowind_bc_ptr;
        this->d_lblock_gid_offsets = d_lblock_gid_offsets;
        this->d_lblock_start_offsets = d_lblock_start_offsets;
    }

    __device__ void operator()(const int_t &i) const
    {   
        if(i == 0)
            d_lblock_gid_offsets[i] = d_lblock_start_offsets[i] = 0;
        else
        {
            int_t *Lrowind_bc = Lrowind_bc_ptr[i - 1];
            d_lblock_gid_offsets[i] = (Lrowind_bc ? Lrowind_bc[0] : 0);
            d_lblock_start_offsets[i] = (Lrowind_bc ? Lrowind_bc[0] + 1 : 0);
        }
    }
};

struct BatchLDataAssign_Func {
    int_t **Lrowind_bc_ptr, **d_lblock_gid_ptrs, **d_lblock_start_ptrs;

    BatchLDataAssign_Func(int_t** Lrowind_bc_ptr, int_t** d_lblock_gid_ptrs, int_t** d_lblock_start_ptrs)
    {
        this->Lrowind_bc_ptr = Lrowind_bc_ptr;
        this->d_lblock_gid_ptrs = d_lblock_gid_ptrs;
        this->d_lblock_start_ptrs = d_lblock_start_ptrs;
    }

    __device__ void operator()(const int_t &i) const
    {   
        int_t *Lrowind_bc = Lrowind_bc_ptr[i];
        if(!Lrowind_bc)
            d_lblock_gid_ptrs[i] = d_lblock_start_ptrs[i] = NULL;
        else
        {   
            int_t *block_gids = d_lblock_gid_ptrs[i], *block_starts = d_lblock_start_ptrs[i];
            int_t nblocks = Lrowind_bc[0], Lptr = BC_HEADER, psum = 0;
            for(int_t b = 0; b < nblocks; b++)
            {
                block_gids[b] = Lrowind_bc[Lptr];
                int_t nrows = Lrowind_bc[Lptr + 1];
                block_starts[b] = psum;
                psum += nrows;
                Lptr += nrows + LB_DESCRIPTOR;
            }
            block_starts[nblocks] = psum;
        }
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Device functions and kernels 
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ inline int_t find_entry_index_flat(int_t *index_list, int_t index, int_t n)
{
    int threadId = threadIdx.x;
    __shared__ int_t idx;
    
    if (!threadId)    
        idx = -1;

    __syncthreads();

    int nThreads = blockDim.x;
    int blocksPerThreads = CEILING(n, nThreads);

    for (int_t blk = blocksPerThreads * threadIdx.x;
         blk < blocksPerThreads * (threadIdx.x + 1);
         blk++)
    {
        if (blk < n)
        {
            if(index == index_list[blk])
                idx = blk;
        }
    }
    __syncthreads();
    return idx;
}

__device__ inline int computeIndirectMapGPU_flat(int_t *rcS2D, int_t srcLen, int_t *srcVec, int_t src_first_index,
                                     int_t dstLen, int_t *dstVec, int_t dst_first_index, int_t *dstIdx)
{
    int threadId = threadIdx.x;
    if (dstVec == NULL) /*uncompressed dimension*/
    {
        if (threadId < srcLen)
            rcS2D[threadId] = srcVec[threadId] - src_first_index;
        __syncthreads();
        return 0;
    }

    if (threadId < dstLen)
        dstIdx[dstVec[threadId] - dst_first_index] = threadId;
    __syncthreads();

    if (threadId < srcLen)
        rcS2D[threadId] = dstIdx[srcVec[threadId] - src_first_index];
    __syncthreads();

    return 0;
}

template<class T>
__global__ void scatterGPU_batch_flat(
    int_t k_st, int_t maxSuperSize, T **gemmBuff_ptrs, BatchDim_t *LDgemmBuff_batch,
    T **Unzval_br_new_ptr, int_t** Ucolind_br_ptr, T** Lnzval_bc_ptr, 
    int_t** Lrowind_bc_ptr, int_t** lblock_gid_ptrs, int_t **lblock_start_ptrs, 
    int_t *dperm_c_supno, int_t *xsup
)
{
    int batch_index = blockIdx.z;
    int_t k = dperm_c_supno[k_st + batch_index];
    
    T* gemmBuff = gemmBuff_ptrs[batch_index];
    int_t *Ucolind_br = Ucolind_br_ptr[k];
    int_t *Lrowind_bc = Lrowind_bc_ptr[k];
    int_t *lblock_gid = lblock_gid_ptrs[k];
    int_t *lblock_start = lblock_start_ptrs[k];

    if(!Ucolind_br || !Lrowind_bc || !gemmBuff || !lblock_gid || !lblock_start)
        return;

    int_t L_blocks = Lrowind_bc[0];    
    int_t U_blocks = Ucolind_br[0];
    BatchDim_t LDgemmBuff = LDgemmBuff_batch[batch_index];

    int_t ii = 1 + blockIdx.x;
    int_t jj = blockIdx.y;

    if(ii >= L_blocks || jj >= U_blocks)
        return;

    // calculate gi, gj
    int threadId = threadIdx.x;

    int_t gi = lblock_gid[ii];
    int_t gj = Ucolind_br[UB_DESCRIPTOR_NEWUCPP + jj];
    
    T *Dst;
    int_t lddst;
    int_t dstRowLen, dstColLen;
    int_t *dstRowList;
    int_t *dstColList;
    int_t dst_row_first_index, dst_col_first_index;
    int_t li = 0, lj = 0;

    if (gj > gi) // its in upanel
    {
        int_t* U_index_i = Ucolind_br_ptr[gi];
        int_t nub = U_index_i[0];
        lddst = U_index_i[2];
        lj = find_entry_index_flat(U_index_i + UB_DESCRIPTOR_NEWUCPP, gj, nub);
        li = gi;
        int_t col_offset = U_index_i[UB_DESCRIPTOR_NEWUCPP + nub + lj];
        Dst = Unzval_br_new_ptr[gi] + lddst * col_offset;
        dstRowLen = lddst;
        dstRowList = NULL;
        dst_row_first_index = 0;
        dstColLen = U_index_i[UB_DESCRIPTOR_NEWUCPP + nub + lj + 1] - col_offset;
        dstColList = U_index_i + UB_DESCRIPTOR_NEWUCPP + 2 * nub + 1 + col_offset;
        dst_col_first_index = xsup[gj];
    }
    else
    {
        int_t* L_index_j = Lrowind_bc_ptr[gj], *ljblock_start = lblock_start_ptrs[gj];
        int_t nlb = L_index_j[0];
        lddst = L_index_j[1];
        li = find_entry_index_flat(lblock_gid_ptrs[gj], gi, nlb);
        lj = gj;
        int_t row_offset = ljblock_start[li];
        Dst = Lnzval_bc_ptr[gj] + row_offset;
        dstRowLen = ljblock_start[li + 1] - row_offset;
        dstRowList = L_index_j + BC_HEADER + (li + 1) * LB_DESCRIPTOR + row_offset;
        dst_row_first_index = xsup[gi];
        dstColLen = SuperSize(gj);
        dstColList = NULL;
        dst_col_first_index = 0;
    }

    // compute source row to dest row mapping
    extern __shared__ int_t baseSharedPtr[];
    int_t *rowS2D = baseSharedPtr;
    int_t *colS2D = &rowS2D[maxSuperSize];
    int_t *dstIdx = &colS2D[maxSuperSize];

    int_t ublock_start = Ucolind_br[UB_DESCRIPTOR_NEWUCPP + U_blocks + jj];
    int_t nrows = lblock_start[ii + 1] - lblock_start[ii];
    int_t ncols = Ucolind_br[UB_DESCRIPTOR_NEWUCPP + U_blocks + jj + 1] - ublock_start;

    int_t *lpanel_row_list = Lrowind_bc + BC_HEADER + (ii + 1) * LB_DESCRIPTOR + lblock_start[ii];
    int_t *upanel_col_list = Ucolind_br + UB_DESCRIPTOR_NEWUCPP + 2 * U_blocks + 1 + ublock_start;
    int_t lpanel_first_index = xsup[gi];
    int_t upanel_first_index = xsup[gj];

    computeIndirectMapGPU_flat(rowS2D, nrows, lpanel_row_list, lpanel_first_index,
                          dstRowLen, dstRowList, dst_row_first_index, dstIdx);

    // compute source col to dest col mapping
    computeIndirectMapGPU_flat(colS2D, ncols, upanel_col_list, upanel_first_index,
                          dstColLen, dstColList, dst_col_first_index, dstIdx);

    int nThreads = blockDim.x;
    int colsPerThreadBlock = nThreads / nrows;

    int_t rowOff = lblock_start[ii] - lblock_start[1];
    int_t colOff = ublock_start;

    T *Src = &gemmBuff[rowOff + colOff * LDgemmBuff];
    int_t ldsrc = LDgemmBuff;

    // TODO: this seems inefficient
    if (threadId < nrows * colsPerThreadBlock)
    {
        /* 1D threads are logically arranged in 2D shape. */
        int i = threadId % nrows;
        int j = threadId / nrows;

#pragma unroll 4
        while (j < ncols)
        {

#define ATOMIC_SCATTER
// Atomic Scatter is need if I want to perform multiple Schur Complement
//  update concurrently
#ifdef ATOMIC_SCATTER
             atomicAddT(&Dst[rowS2D[i] + lddst * colS2D[j]], -Src[i + ldsrc * j]);
#else
            Dst[rowS2D[i] + lddst * colS2D[j]] -= Src[i + ldsrc * j];
#endif
            j += colsPerThreadBlock;
        }
    }

    __syncthreads();
}

template<class T>
inline void scatterGPU_batchDriver_flat(
    int_t k_st, int_t maxSuperSize, T **gemmBuff_ptrs, BatchDim_t *LDgemmBuff_batch,
    T **Unzval_br_new_ptr, int_t** Ucolind_br_ptr, T** Lnzval_bc_ptr, 
    int_t** Lrowind_bc_ptr, int_t** lblock_gid_ptrs, int_t **lblock_start_ptrs, 
    int_t *dperm_c_supno, int_t *xsup, int_t ldt, BatchDim_t max_ilen, BatchDim_t max_jlen, 
    BatchDim_t batchCount, hipStream_t cuStream
)
{
    const BatchDim_t op_increment = 65535;
    
    for(BatchDim_t op_start = 0; op_start < batchCount; op_start += op_increment)
	{
		BatchDim_t batch_size = std::min(op_increment, batchCount - op_start);
    
        dim3 dimBlock(ldt); // 1d thread
        dim3 dimGrid(max_ilen, max_jlen, batch_size);
        size_t sharedMemorySize = 3 * maxSuperSize * sizeof(int_t);

        scatterGPU_batch_flat<<<dimGrid, dimBlock, sharedMemorySize, cuStream>>>(
            k_st + op_start, maxSuperSize, gemmBuff_ptrs, LDgemmBuff_batch, Unzval_br_new_ptr,
            Ucolind_br_ptr, Lnzval_bc_ptr, Lrowind_bc_ptr, lblock_gid_ptrs, lblock_start_ptrs, 
            dperm_c_supno, xsup
        );
    }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<class T>
void computeLBlockData(TBatchFactorizeWorkspace<T>* ws, int_t nsupers)
{
    LocalLU_type<T>& d_localLU = ws->d_localLU;

    // Allocate memory for the offsets and the pointers 
    gpuErrchk( hipMalloc(&ws->d_lblock_gid_offsets, sizeof(int64_t) * (nsupers + 1)) );
    gpuErrchk( hipMalloc(&ws->d_lblock_start_offsets, sizeof(int64_t) * (nsupers + 1)) );
    gpuErrchk( hipMalloc(&ws->d_lblock_gid_ptrs, sizeof(int_t*) * nsupers) );
    gpuErrchk( hipMalloc(&ws->d_lblock_start_ptrs, sizeof(int_t*) * nsupers) );

    // Initialize to the block counts for each panel
    thrust::for_each(
        thrust::system::cuda::par, thrust::counting_iterator<int_t>(0), 
        thrust::counting_iterator<int_t>(nsupers + 1), BatchLDataSizeAssign_Func(
            d_localLU.Lrowind_bc_ptr, ws->d_lblock_gid_offsets, ws->d_lblock_start_offsets
    ) );
    
    // Do an inclusive scan to compute offsets and get the total amount of blocks 
    ws->total_l_blocks = *(thrust::device_ptr<int64_t>(
        thrust::inclusive_scan(
            thrust::system::cuda::par, ws->d_lblock_gid_offsets + 1, 
            ws->d_lblock_gid_offsets + nsupers + 1, ws->d_lblock_gid_offsets + 1
    ) ) - 1);

    ws->total_start_size = *(thrust::device_ptr<int64_t>(
        thrust::inclusive_scan(
            thrust::system::cuda::par, ws->d_lblock_start_offsets + 1, 
            ws->d_lblock_start_offsets + nsupers + 1, ws->d_lblock_start_offsets + 1
    ) ) - 1);

    // Allocate the block data 
    gpuErrchk( hipMalloc(&ws->d_lblock_gid_dat, sizeof(int_t) * ws->total_l_blocks) );
    gpuErrchk( hipMalloc(&ws->d_lblock_start_dat, sizeof(int_t) * ws->total_start_size) );

    // Generate the pointers 
    generateOffsetPointers(ws->d_lblock_gid_dat, ws->d_lblock_gid_offsets, ws->d_lblock_gid_ptrs, nsupers);
    generateOffsetPointers(ws->d_lblock_start_dat, ws->d_lblock_start_offsets, ws->d_lblock_start_ptrs, nsupers);

    // Now copy the data over from d_localLU
    thrust::for_each(
        thrust::system::cuda::par, thrust::counting_iterator<int_t>(0), 
        thrust::counting_iterator<int_t>(nsupers), BatchLDataAssign_Func(
        d_localLU.Lrowind_bc_ptr, ws->d_lblock_gid_ptrs, ws->d_lblock_start_ptrs
    ) );
}

template<class T>
void batchAllocateGemmBuffers(
    TBatchFactorizeWorkspace<T>* ws, LUStruct_type<T> *LUstruct, trf3dpartitionType<T> *trf3Dpartition, 
    gridinfo3d_t *grid3d
)
{
    int_t mxLeafNode = trf3Dpartition->mxLeafNode;

    // TODO: is this necessary if this is being done on a single node?
    int_t maxLvl = log2i(grid3d->zscp.Np) + 1;

    std::vector<int64_t> gemmCsizes(mxLeafNode, 0);
	int_t mx_fsize = 0;
	
	for (int_t ilvl = 0; ilvl < maxLvl; ++ilvl) 
    {
	    int_t treeId = trf3Dpartition->myTreeIdxs[ilvl];
	    sForest_t* sforest = trf3Dpartition->sForests[treeId];
	    if (sforest)
        {
            int_t *perm_c_supno = sforest->nodeList;
            mx_fsize = max(mx_fsize, sforest->nNodes);

            int_t maxTopoLevel = sforest->topoInfo.numLvl;
            for (int_t topoLvl = 0; topoLvl < maxTopoLevel; ++topoLvl) 
            {
                int_t k_st = sforest->topoInfo.eTreeTopLims[topoLvl];
                int_t k_end = sforest->topoInfo.eTreeTopLims[topoLvl + 1];
            
                for (int_t k0 = k_st; k0 < k_end; ++k0) 
                {
                    int_t offset = k0 - k_st;
                    int_t k = perm_c_supno[k0];
                    int_t* L_data = LUstruct->Llu->Lrowind_bc_ptr[k];
                    int_t* U_data = LUstruct->Llu->Ucolind_br_ptr[k];
                    if(L_data && U_data)
                    {
                        int_t Csize = L_data[1] * U_data[1];
                        gemmCsizes[offset] = SUPERLU_MAX(gemmCsizes[offset], Csize);
                    }
                }
		    }
	    }
	}
    // Allocate the gemm buffers 
    gpuErrchk( hipMalloc(&(ws->gemm_buff_ptrs), sizeof(T*) * mxLeafNode) );
    gpuErrchk( hipMalloc(&(ws->gemm_buff_offsets), sizeof(int64_t) * (mxLeafNode + 1)) );

    // Copy the host offset to the device 
    gpuErrchk( hipMemcpy( ws->gemm_buff_offsets + 1, gemmCsizes.data(), mxLeafNode * sizeof(int64_t), hipMemcpyHostToDevice) );
    *(thrust::device_ptr<int64_t>(ws->gemm_buff_offsets)) = 0;

    int64_t total_entries = *(thrust::device_ptr<int64_t>(
        thrust::inclusive_scan(
            thrust::system::cuda::par, ws->gemm_buff_offsets + 1, 
            ws->gemm_buff_offsets + mxLeafNode + 1, ws->gemm_buff_offsets + 1
    ) ) - 1);

    // Allocate the base memory and generate the pointers on the device
    gpuErrchk(hipMalloc(&(ws->gemm_buff_base), sizeof(T) * total_entries));
    generateOffsetPointers(ws->gemm_buff_base, ws->gemm_buff_offsets, ws->gemm_buff_ptrs, mxLeafNode);

    // Allocate GPU copy for the node list 
    gpuErrchk(hipMalloc(&(ws->perm_c_supno), sizeof(int_t) * mx_fsize));
}

template<class T>
void copyHostLUDataToGPU(TBatchFactorizeWorkspace<T>* ws, LocalLU_type<T>* host_Llu, int_t nsupers)
{
    LocalLU_type<T>& d_localLU = ws->d_localLU;

    // Allocate data, offset and ptr arrays for the indices and lower triangular blocks 
    d_localLU.Lrowind_bc_cnt = host_Llu->Lrowind_bc_cnt;
    gpuErrchk( hipMalloc(&(d_localLU.Lrowind_bc_dat), d_localLU.Lrowind_bc_cnt * sizeof(int_t)) );
    gpuErrchk( hipMalloc(&(d_localLU.Lrowind_bc_offset), nsupers * sizeof(long int)) );
    gpuErrchk( hipMalloc(&(d_localLU.Lrowind_bc_ptr), nsupers * sizeof(int_t*)) );

    d_localLU.Lnzval_bc_cnt = host_Llu->Lnzval_bc_cnt;
    gpuErrchk( hipMalloc(&(d_localLU.Lnzval_bc_dat), d_localLU.Lnzval_bc_cnt * sizeof(T)) );
    gpuErrchk( hipMalloc(&(d_localLU.Lnzval_bc_offset), nsupers * sizeof(long int)) );
    gpuErrchk( hipMalloc(&(d_localLU.Lnzval_bc_ptr), nsupers * sizeof(T*)) );

    // Allocate data, offset and ptr arrays for the indices and upper triangular blocks 
    d_localLU.Ucolind_br_cnt = host_Llu->Ucolind_br_cnt;
    gpuErrchk( hipMalloc(&(d_localLU.Ucolind_br_dat), d_localLU.Ucolind_br_cnt * sizeof(int_t)) );
    gpuErrchk( hipMalloc(&(d_localLU.Ucolind_br_offset), nsupers * sizeof(int64_t)) );
    gpuErrchk( hipMalloc(&(d_localLU.Ucolind_br_ptr), nsupers * sizeof(int_t*)) );

    d_localLU.Unzval_br_new_cnt = host_Llu->Unzval_br_new_cnt;
    gpuErrchk( hipMalloc(&(d_localLU.Unzval_br_new_dat), d_localLU.Unzval_br_new_cnt * sizeof(T)) );
    gpuErrchk( hipMalloc(&(d_localLU.Unzval_br_new_offset), nsupers * sizeof(int64_t)) );
    gpuErrchk( hipMalloc(&(d_localLU.Unzval_br_new_ptr), nsupers * sizeof(T*)) );

    // Copy the index and nzval data over to the GPU 
    gpuErrchk( hipMemcpy(d_localLU.Lrowind_bc_dat, host_Llu->Lrowind_bc_dat, d_localLU.Lrowind_bc_cnt * sizeof(int_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_localLU.Lrowind_bc_offset, host_Llu->Lrowind_bc_offset, nsupers * sizeof(long int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_localLU.Lnzval_bc_dat, host_Llu->Lnzval_bc_dat, d_localLU.Lnzval_bc_cnt * sizeof(T), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_localLU.Lnzval_bc_offset, host_Llu->Lnzval_bc_offset, nsupers * sizeof(long int), hipMemcpyHostToDevice) );
    
    gpuErrchk( hipMemcpy(d_localLU.Ucolind_br_dat, host_Llu->Ucolind_br_dat, d_localLU.Ucolind_br_cnt * sizeof(int_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_localLU.Ucolind_br_offset, host_Llu->Ucolind_br_offset, nsupers * sizeof(int64_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_localLU.Unzval_br_new_dat, host_Llu->Unzval_br_new_dat, d_localLU.Unzval_br_new_cnt * sizeof(T), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_localLU.Unzval_br_new_offset, host_Llu->Unzval_br_new_offset, nsupers * sizeof(int64_t), hipMemcpyHostToDevice) );
    
    // Generate the pointers using the offsets 
    generateOffsetPointers(d_localLU.Lrowind_bc_dat, d_localLU.Lrowind_bc_offset, d_localLU.Lrowind_bc_ptr, nsupers);
    generateOffsetPointers(d_localLU.Lnzval_bc_dat, d_localLU.Lnzval_bc_offset, d_localLU.Lnzval_bc_ptr, nsupers);
    generateOffsetPointers(d_localLU.Ucolind_br_dat, d_localLU.Ucolind_br_offset, d_localLU.Ucolind_br_ptr, nsupers);
    generateOffsetPointers(d_localLU.Unzval_br_new_dat, d_localLU.Unzval_br_new_offset, d_localLU.Unzval_br_new_ptr, nsupers);

    // Copy the L data for global ids and block offsets into a more parallel friendly data structure 
    computeLBlockData(ws, nsupers);
}

template<class T>
void TFactBatchSolve(TBatchFactorizeWorkspace<T>* ws, int_t k_st, int_t k_end)
{
#ifdef HAVE_MAGMA
    LocalLU_type<T>& d_localLU = ws->d_localLU;
    TBatchLUMarshallData<T>& mdata = ws->marshall_data;
    TBatchSCUMarshallData<T>& sc_mdata = ws->sc_marshall_data;

    const T t_one = one<T>(), t_zero = zeroT<T>();

    // Diagonal block batched LU decomposition   
    marshallBatchedLUData<T>(ws, k_st, k_end);
    
    int_t info = magma_getrf_nopiv_vbatched(
        mdata.dev_diag_dim_array, mdata.dev_diag_dim_array, 
        mdata.dev_diag_ptrs, mdata.dev_diag_ld_array, 
        NULL, mdata.dev_info_array, mdata.batchsize, 
        ws->magma_queue
    );
    
    int max_info = thrust::reduce(thrust::system::cuda::par, mdata.dev_info_array, mdata.dev_info_array + mdata.batchsize, 0, thrust::maximum<BatchDim_t>());
    printf("Factor info = %d max_info = %d\n", info, max_info);

    // Upper panel batched triangular solves
    marshallBatchedTRSMUData<T>(ws, k_st, k_end);

    magmablas_trsm_vbatched_nocheck(
        MagmaLeft, MagmaLower, MagmaNoTrans, MagmaUnit, 
        mdata.dev_diag_dim_array, mdata.dev_panel_dim_array, t_one, 
        mdata.dev_diag_ptrs, mdata.dev_diag_ld_array, 
        mdata.dev_panel_ptrs, mdata.dev_panel_ld_array, 
        mdata.batchsize, ws->magma_queue
    );

    // Lower panel batched triangular solves
    marshallBatchedTRSMLData<T>(ws, k_st, k_end);

    magmablas_trsm_vbatched_nocheck(
        MagmaRight, MagmaUpper, MagmaNoTrans, MagmaNonUnit, 
        mdata.dev_panel_dim_array, mdata.dev_diag_dim_array, t_one, 
        mdata.dev_diag_ptrs, mdata.dev_diag_ld_array, 
        mdata.dev_panel_ptrs, mdata.dev_panel_ld_array, 
        mdata.batchsize, ws->magma_queue
    );

    // Batched schur complement updates 
    marshallBatchedSCUData<T>(ws, k_st, k_end);
    
    magmablas_gemm_vbatched_max_nocheck (
        MagmaNoTrans, MagmaNoTrans, sc_mdata.dev_m_array, sc_mdata.dev_n_array, sc_mdata.dev_k_array,
        t_one, sc_mdata.dev_A_ptrs, sc_mdata.dev_lda_array, sc_mdata.dev_B_ptrs, sc_mdata.dev_ldb_array,
        t_zero, sc_mdata.dev_C_ptrs, sc_mdata.dev_ldc_array, sc_mdata.batchsize,
        sc_mdata.max_m, sc_mdata.max_n, sc_mdata.max_k, ws->magma_queue
    );
    
    scatterGPU_batchDriver_flat<T>(
        k_st, ws->maxSuperSize, sc_mdata.dev_C_ptrs, sc_mdata.dev_ldc_array,
        d_localLU.Unzval_br_new_ptr, d_localLU.Ucolind_br_ptr, d_localLU.Lnzval_bc_ptr, 
        d_localLU.Lrowind_bc_ptr, ws->d_lblock_gid_ptrs, ws->d_lblock_start_ptrs, 
        ws->perm_c_supno, ws->xsup, ws->ldt, sc_mdata.max_ilen, sc_mdata.max_jlen, 
        sc_mdata.batchsize, ws->stream
    );
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Main factorization routiunes
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<class T>
int sparseTreeFactorBatchGPUT(TBatchFactorizeWorkspace<T>* ws, sForest_t *sforest)
{
    int_t nnodes = sforest->nNodes; 

    if(nnodes < 1)
        return 1;
    
    // Host list of nodes in the order of factorization copied to the GPU 
    int_t *perm_c_supno = sforest->nodeList; 
    gpuErrchk(hipMemcpy(ws->perm_c_supno, perm_c_supno, sizeof(int_t) * nnodes, hipMemcpyHostToDevice));

    // Tree containing the supernode limits per level 
    treeTopoInfo_t *treeTopoInfo = &sforest->topoInfo;
    int_t maxTopoLevel = treeTopoInfo->numLvl;
    int_t *eTreeTopLims = treeTopoInfo->eTreeTopLims;

    for(int_t topoLvl = 0; topoLvl < maxTopoLevel; topoLvl++)
        TFactBatchSolve<T>(ws, eTreeTopLims[topoLvl], eTreeTopLims[topoLvl + 1]);

    return 0;
}

template<class T>
TBatchFactorizeWorkspace<T>* getBatchFactorizeWorkspaceT(
    int_t nsupers, int_t ldt, trf3dpartitionType<T> *trf3Dpartition, LUStruct_type<T> *LUstruct, 
    gridinfo3d_t *grid3d, superlu_dist_options_t *options, SuperLUStat_t *stat, int *info
)
{
#ifdef HAVE_MAGMA
    TBatchFactorizeWorkspace<T>* ws = new TBatchFactorizeWorkspace<T>();
    
    int device_id;
    gpuErrchk( hipGetDevice(&device_id) );

    int_t* xsup = LUstruct->Glu_persist->xsup;
    int_t n = xsup[nsupers];
    gridinfo_t *grid = &(grid3d->grid2d);

    double tic = SuperLU_timer_();

    pconvert_flatten_skyline2UROWDATA(options, grid, LUstruct, stat, n);

    double convert_time = SuperLU_timer_() - tic;

    // TODO: determine if ldt is supposed to be the same as maxSuperSize?
    ws->ldt = ws->maxSuperSize = ldt;
    ws->nsupers = nsupers;

    // Set up device handles 
    gpuErrchk( hipStreamCreate(&ws->stream) );
    hipblasCreate( &ws->cuhandle );
    magma_queue_create_from_cuda(device_id, ws->stream, ws->cuhandle, NULL, &ws->magma_queue);

    // Copy the xsup to the GPU 
    tic = SuperLU_timer_();
    gpuErrchk(hipMalloc(&ws->xsup, (nsupers + 1) * sizeof(int_t)));
    gpuErrchk(hipMemcpy(ws->xsup, xsup, (nsupers + 1) * sizeof(int_t), hipMemcpyHostToDevice));

    // Copy the flattened LU data over to the GPU 
    // TODO: I currently have to make a GPU friendly copy of the globa ids of blocks within L
    // and compute block offsets. Can this be avoided with a change to the L index structure?
    copyHostLUDataToGPU<T>(ws, LUstruct->Llu, nsupers);

    double copy_time = SuperLU_timer_() - tic;

    // Allocate marhsalling workspace
    tic = SuperLU_timer_();
    ws->marshall_data.setBatchSize(trf3Dpartition->mxLeafNode);
    ws->sc_marshall_data.setBatchSize(trf3Dpartition->mxLeafNode);

    // Determine buffer sizes for schur complement updates and supernode lists 
    batchAllocateGemmBuffers<T>(ws, LUstruct, trf3Dpartition, grid3d);
    double ws_time = SuperLU_timer_() - tic;
    
    printf("\tSky2UROWDATA Convert time = %.4f\n", convert_time);
    printf("\tH2D Copy time = %.4f\n", copy_time);
    printf("\tWorkspace alloc time = %.4f\n", ws_time);

    return ws;
#endif
}

template<class T>
void copyGPULUDataToHostT(
    TBatchFactorizeWorkspace<T>* ws, LUStruct_type<T> *LUstruct, gridinfo3d_t *grid3d,
    SCT_t *SCT_, superlu_dist_options_t *options, SuperLUStat_t *stat
)
{
    LocalLU_type<T>& d_localLU = ws->d_localLU;
    LocalLU_type<T>* host_Llu = LUstruct->Llu;

    double tic = SuperLU_timer_();
    
    // Only need to copy the nzval data arrays when moving from the GPU to the Host 
    gpuErrchk( hipMemcpy(host_Llu->Lnzval_bc_dat, d_localLU.Lnzval_bc_dat, d_localLU.Lnzval_bc_cnt * sizeof(T), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(host_Llu->Unzval_br_new_dat, d_localLU.Unzval_br_new_dat, d_localLU.Unzval_br_new_cnt * sizeof(T), hipMemcpyDeviceToHost) );
    
    double copy_time = SuperLU_timer_() - tic;

    // Convert the host data from block row to skyline 
    int_t* xsup = LUstruct->Glu_persist->xsup;
    int_t n = xsup[ws->nsupers];
    gridinfo_t *grid = &(grid3d->grid2d);

    tic = SuperLU_timer_();
    pconvertUROWDATA2skyline(options, grid, LUstruct, stat, n);
    double convert_time = SuperLU_timer_() - tic;

    printf("\tD2H Copy time = %.4f\n", copy_time);
    printf("\tConvert time = %.4f\n", convert_time);
}

template<class T>
void freeBatchFactorizeWorkspaceT(TBatchFactorizeWorkspace<T>* ws)
{
    gpuErrchk( hipFree(ws->d_lblock_gid_dat) );
    gpuErrchk( hipFree(ws->d_lblock_gid_offsets) );
    gpuErrchk( hipFree(ws->d_lblock_gid_ptrs) );
    gpuErrchk( hipFree(ws->d_lblock_start_dat) );
    gpuErrchk( hipFree(ws->d_lblock_start_offsets) );
    gpuErrchk( hipFree(ws->d_lblock_start_ptrs) );
    gpuErrchk( hipFree(ws->gemm_buff_base) );
    gpuErrchk( hipFree(ws->gemm_buff_offsets) );
    gpuErrchk( hipFree(ws->gemm_buff_ptrs) );
    gpuErrchk( hipFree(ws->perm_c_supno) );
    gpuErrchk( hipFree(ws->xsup) );

    LocalLU_type<T>& d_localLU = ws->d_localLU;
    gpuErrchk( hipFree(d_localLU.Lrowind_bc_dat) );
    gpuErrchk( hipFree(d_localLU.Lrowind_bc_offset) );
    gpuErrchk( hipFree(d_localLU.Lrowind_bc_ptr) );
    gpuErrchk( hipFree(d_localLU.Lnzval_bc_dat) );
    gpuErrchk( hipFree(d_localLU.Lnzval_bc_offset) );
    gpuErrchk( hipFree(d_localLU.Lnzval_bc_ptr) );
    gpuErrchk( hipFree(d_localLU.Ucolind_br_dat) );
    gpuErrchk( hipFree(d_localLU.Ucolind_br_offset) );
    gpuErrchk( hipFree(d_localLU.Ucolind_br_ptr) );
    gpuErrchk( hipFree(d_localLU.Unzval_br_new_dat) );
    gpuErrchk( hipFree(d_localLU.Unzval_br_new_offset) );
    gpuErrchk( hipFree(d_localLU.Unzval_br_new_ptr) );
#ifdef HAVE_MAGMA
    magma_queue_destroy(ws->magma_queue);
#endif
    hipblasDestroy( ws->cuhandle );
    gpuErrchk( hipStreamDestroy(ws->stream) );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// C interface
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
typedef TBatchFactorizeWorkspace<float >        sBatchFactorizeWorkspace;
typedef TBatchFactorizeWorkspace<double>        dBatchFactorizeWorkspace;
typedef TBatchFactorizeWorkspace<doublecomplex> zBatchFactorizeWorkspace;

extern "C" {
//single   
int ssparseTreeFactorBatchGPU(sBatchFactorizeWorkspace* ws, sForest_t *sforest)
{
    return sparseTreeFactorBatchGPUT<float>(ws, sforest);
}

sBatchFactorizeWorkspace* sgetBatchFactorizeWorkspace(
    int_t nsupers, int_t ldt, strf3Dpartition_t *trf3Dpartition, sLUstruct_t *LUstruct, 
    gridinfo3d_t *grid3d, superlu_dist_options_t *options, SuperLUStat_t *stat, int *info
)
{ 
    return getBatchFactorizeWorkspaceT<float>(nsupers, ldt, trf3Dpartition, LUstruct, grid3d, options, stat, info); 
}

void scopyGPULUDataToHost(
    sBatchFactorizeWorkspace* ws, sLUstruct_t *LUstruct, gridinfo3d_t *grid3d,
    SCT_t *SCT_, superlu_dist_options_t *options, SuperLUStat_t *stat
)
{ 
    copyGPULUDataToHostT<float>(ws, LUstruct, grid3d, SCT_, options, stat); 
}

void sfreeBatchFactorizeWorkspace(sBatchFactorizeWorkspace* ws)
{ 
    freeBatchFactorizeWorkspaceT<float>(ws); 
}

//double  
int dsparseTreeFactorBatchGPU(dBatchFactorizeWorkspace* ws, sForest_t *sforest)
{
    return sparseTreeFactorBatchGPUT<double>(ws, sforest);
}

dBatchFactorizeWorkspace* dgetBatchFactorizeWorkspace(
    int_t nsupers, int_t ldt, dtrf3Dpartition_t *trf3Dpartition, dLUstruct_t *LUstruct, 
    gridinfo3d_t *grid3d, superlu_dist_options_t *options, SuperLUStat_t *stat, int *info
)
{ 
    return getBatchFactorizeWorkspaceT<double>(nsupers, ldt, trf3Dpartition, LUstruct, grid3d, options, stat, info); 
}

void dcopyGPULUDataToHost(
    dBatchFactorizeWorkspace* ws, dLUstruct_t *LUstruct, gridinfo3d_t *grid3d,
    SCT_t *SCT_, superlu_dist_options_t *options, SuperLUStat_t *stat
)
{ 
    copyGPULUDataToHostT<double>(ws, LUstruct, grid3d, SCT_, options, stat); 
}

void dfreeBatchFactorizeWorkspace(dBatchFactorizeWorkspace* ws)
{ 
    freeBatchFactorizeWorkspaceT<double>(ws); 
}

//doublecomplex 
int zsparseTreeFactorBatchGPU(zBatchFactorizeWorkspace* ws, sForest_t *sforest)
{
    return sparseTreeFactorBatchGPUT<doublecomplex>(ws, sforest);
}

zBatchFactorizeWorkspace* zgetBatchFactorizeWorkspace(
    int_t nsupers, int_t ldt, ztrf3Dpartition_t *trf3Dpartition, zLUstruct_t *LUstruct, 
    gridinfo3d_t *grid3d, superlu_dist_options_t *options, SuperLUStat_t *stat, int *info
)
{ 
    return getBatchFactorizeWorkspaceT<doublecomplex>(nsupers, ldt, trf3Dpartition, LUstruct, grid3d, options, stat, info); 
}

void zcopyGPULUDataToHost(
    zBatchFactorizeWorkspace* ws, zLUstruct_t *LUstruct, gridinfo3d_t *grid3d,
    SCT_t *SCT_, superlu_dist_options_t *options, SuperLUStat_t *stat
)
{ 
    copyGPULUDataToHostT<doublecomplex>(ws, LUstruct, grid3d, SCT_, options, stat); 
}

void zfreeBatchFactorizeWorkspace(zBatchFactorizeWorkspace* ws)
{ 
    freeBatchFactorizeWorkspaceT<doublecomplex>(ws); 
}

}